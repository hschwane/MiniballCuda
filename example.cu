#include "hip/hip_runtime.h"
/*
 * MiniballCuda
 * main.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Copyright (c) 2020 Hendrik Schwanekamp
 *
 */

#include <random>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime_api.h>

#include "MiniballCuda.hpp"

// ----------------
// settings
constexpr int d = 3; // dimensions need to be constexpr
constexpr int n = 6; // n of a single ball can be lower, but never higher than this
int numOfSpheres = 100000; // number of spheres to build
constexpr int iterations = 10; // number of iterations to average timings
// ----------------

__global__ void miniballExampleKernel(int numSpheres, const float* input, float* output)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= numSpheres)
        return;

    // load data from global memory into local buffer
    int startId = d*n*index; // every thread processes one sphere
    float data[n][d];
    for (int i=0; i<n; ++i)
        for (int j=0; j<d; ++j)
            data[i][j] = input[startId + i*d + j];

    // define the types of iterators through the points and their coordinates
    // ----------------------------------------------------------------------
    typedef const float (*PointIterator)[d];
    typedef const float* CoordIterator;

    // create an instance of MiniballCuda
    // ------------------------------
    typedef MiniballCuda::Miniball <MiniballCuda::CoordAccessor<PointIterator, CoordIterator>,d,n,false> MB;
    MB mb( &data[0], &data[n]);

    // store center and radii in output buffer
    auto center = mb.center();
    for(int i=0; i<d; ++i, ++center)
        output[index*(d+1) +i] = *center;
    output[index*(d+1) +d] = mb.squared_radius();
}

void checkCudaError(hipError_t code)
{
    if(code != hipSuccess)
        throw std::runtime_error("Cuda error:" + std::string(hipGetErrorString(code)));
}

int main()
{
    // get some cuda memory
    float* input;
    float* output;

    checkCudaError( hipMallocManaged(&input, d*n*numOfSpheres* sizeof(float)) );
    checkCudaError( hipMallocManaged(&output, (d+1)*numOfSpheres* sizeof(float)) );

    // generate some random input
    std::random_device rd;
    std::default_random_engine rng(rd());
    std::uniform_real_distribution<float> dist(0,1);

    for(int i =0; i<d*n*numOfSpheres; i++)
        input[i] = dist(rng);

    // ------------------------------------------------------------
    // now call the cuda kernel
    int device = -1;
    hipGetDevice(&device);
    hipMemPrefetchAsync(input, d*n*numOfSpheres*sizeof(float), device, NULL);
    hipMemPrefetchAsync(output, (d+1)*numOfSpheres*sizeof(float), device, NULL);

    int blockSize = 256; // experiment with different block sizes depending on your GPU
    int numBlocks = (numOfSpheres + blockSize - 1) / blockSize;

    auto startTime = std::chrono::steady_clock::now();
    for(int j =0; j<iterations; j++)
    {
        miniballExampleKernel<<<numBlocks,blockSize>>>(numOfSpheres,input,output);
        checkCudaError(hipGetLastError());
        checkCudaError(hipDeviceSynchronize());
    }
    std::cout << "Duration GPU: "
              << std::chrono::duration_cast<std::chrono::duration<double>>(std::chrono::steady_clock::now() - startTime).count()*1000.0/iterations
              << "ms" << std::endl;

    // free memory
    hipFree(input);
    hipFree(output);
}